
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__host__ __device__ static __inline__ hipDoubleComplex cuCsqr(hipDoubleComplex a) {
	// Square a complex number
	hipDoubleComplex result;
	result = make_hipDoubleComplex((hipCreal(a) * hipCreal(a)) -
							(hipCimag(a) * hipCimag(a)),
							2.0 * hipCreal(a) * hipCimag(a));
	return result;
}
