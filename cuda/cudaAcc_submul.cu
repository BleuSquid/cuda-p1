
#include <hip/hip_runtime.h>
__global__ void sub_mul_kernel(double *g_out, double *a, double *b1, double *b2, double *ct, int n) {
	const int j2 = blockIdx.x * blockDim.x + threadIdx.x;
	double wkr, wki, xr, xi, yr, yi, ajr, aji, akr, aki, bjr, bji, bkr, bki;
	double new_ajr, new_aji, new_akr, new_aki;
	const int m = n >> 1;
	const int nc = n >> 2;
	const int j = j2 << 1;

	if (j2) {
		int nminusj = n - j;

		wkr = 0.5 - ct[nc - j2];
		wki = ct[j2];

		ajr = a[j];
		aji = a[1 + j];
		akr = a[nminusj];
		aki = a[1 + nminusj];
		xr = ajr - akr;
		xi = aji + aki;
		yr = wkr * xr - wki * xi;
		yi = wkr * xi + wki * xr;
		ajr -= yr;
		aji -= yi;
		akr += yr;
		aki -= yi;

		bjr = b1[j] - b2[j];
		bji = b1[1 + j] - b2[1 + j];
		bkr = b1[nminusj] - b2[nminusj];
		bki = b1[1 + nminusj] - b2[1 + nminusj];

		new_aji = ajr * bji + bjr * aji;
		new_ajr = ajr * bjr - aji * bji;
		new_aki = akr * bki + bkr * aki;
		new_akr = akr * bkr - aki * bki;

		xr = new_ajr - new_akr;
		xi = new_aji + new_aki;
		yr = wkr * xr + wki * xi;
		yi = wkr * xi - wki * xr;
		g_out[j] = new_ajr - yr;
		g_out[1 + j] = yi - new_aji;
		g_out[nminusj] = new_akr + yr;
		g_out[1 + nminusj] = yi - new_aki;
	} else {
		xr = a[0];
		xi = a[1];
		yr = b1[0] - b2[0];
		yi = b1[1] - b2[1];
		g_out[0] = xr * yr + xi * yi;
		g_out[1] = -xr * yi - xi * yr;
		xr = a[0 + m];
		xi = a[1 + m];
		yr = b1[0 + m] - b2[0 + m];
		yi = b1[1 + m] - b2[1 + m];
		g_out[1 + m] = -xr * yi - xi * yr;
		g_out[0 + m] = xr * yr - xi * yi;
	}
}

void cudaAcc_sub_mul(int threads, int n, double *g_out, double *a, double *b1, double *b2, double *ct) {
	sub_mul_kernel<<<n / (4 * threads), threads>>>(g_out, a, b1, b2, ct, n);
}
